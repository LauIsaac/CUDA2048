#include "hip/hip_runtime.h"

#include <wb.h>
#include "types.h"
#include <stdio.h>
#include <stdbool.h>
#include <hiprand/hiprand_kernel.h>


#define m0Mask 0xC000
#define m1Mask 0x3000
#define m2Mask 0x0C00
#define m3Mask 0x0300
#define m4Mask 0x00C0
#define m5Mask 0x0030
#define m6Mask 0x000C
#define m7Mask 0x0003



#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)







/**
 * This function returns a score for the Board. Right now just returns the game score, may be worked to include neighboring tiles combined scores if we need more accuracy.
 * @param input Pointer to Board to be scored.
 * @return The score
 */
__device__ uint32_t score(Board * input){
    uint32_t scoreVal;
    scoreVal = 0;
    for(uint8_t i=0; i < HEIGHT; i++){
        for(uint8_t j=0; j < WIDTH; j++){
            if((*input)[i][j] == 2){
                scoreVal += 2;
            }
            else if((*input)[i][j] != 0){
                scoreVal += (*input)[i][j]+(((*input)[i][j])/2);
            }
            printf("[%d] ",(*input)[i][j]);
        }
        printf("\r\n");
    }
    return scoreVal;
}

__device__ void leftSolver(Board * output){
    int8_t i, j, moveCounter, mergeCounter;

    //This section moves all items through the 0's.
    moveCounter = 0;
    for(i=0; i < HEIGHT; i++){
        for(j=0; j < WIDTH; j++){
            int test = (*output)[i][j];
            if(test == 0){
                moveCounter++;
            }
            else if(moveCounter != 0){
                ((*output))[i][(j-moveCounter)] = ((*output))[i][j];
                ((*output))[i][j] = 0;
            }
        }
        moveCounter = 0;
    }

    //This section merges any nearby values
    mergeCounter = 0;
    for(i=0; i < HEIGHT; i++){
        for(j=0; j < WIDTH-1; j++){
            if(((*output))[i][j] == ((*output))[i][j+1]){
                ((*output))[i][j] = 2*(((*output))[i][j]);
                mergeCounter++;
                ((*output))[i][j+1] = 0;
            }
            if(mergeCounter != 0 && ((*output))[i][j+1] != 0){
                (*output)[i][j] = (*output)[i][j+1];
                (*output)[i][j+1] = 0;
            }
        }
        mergeCounter = 0;
    }



}


__device__ void rightSolver(Board * output){
    int8_t i, j, moveCounter, mergeCounter;

    //This section moves all items through the 0's.
    moveCounter = 0;
    for(i=0; i < HEIGHT; i++){
        for(j=WIDTH-1; j >= 0; j--){
            int test = (*output)[i][j];
            if(test == 0){
                moveCounter++;
            }
            else if(moveCounter != 0){
                ((*output))[i][(j+moveCounter)] = ((*output))[i][j];
                ((*output))[i][j] = 0;
            }
        }
        moveCounter = 0;
    }

    //This section merges any nearby values
    mergeCounter = 0;
    for(i=0; i < HEIGHT; i++){
        for(j=WIDTH-1; j > 0; j--){
            if(((*output))[i][j] == ((*output))[i][j-1]){
                ((*output))[i][j] = 2*(((*output))[i][j]);
                mergeCounter++;
                ((*output))[i][j-1] = 0;
            }
            if(mergeCounter != 0 && (*output)[i][j-1] != 0){
                (*output)[i][j] = (*output)[i][j-1];
                (*output)[i][j-1] = 0;
            }
        }
        mergeCounter = 0;
    }
}


__device__ void upSolver(Board * output){
    int8_t i, j, moveCounter, mergeCounter;

    //This section moves all items through the 0's.
    moveCounter = 0;
    for(j=0; j < WIDTH; j++){
        for(i=0; i < HEIGHT; i++){
            if((*output)[i][j] == 0){
                moveCounter++;
            }
            else if(moveCounter != 0){
                (*output)[(i-moveCounter)][j] = (*output)[i][j];
                (*output)[i][j] = 0;
            }
        }
        moveCounter = 0;
    }

    mergeCounter = 0;
    for(j=0; j < WIDTH; j++){
        for(i=0; i < HEIGHT-1; i++){
            if((*output)[i][j] == (*output)[i+1][j]){
                (*output)[i][j] = 2*((*output)[i][j]);
                mergeCounter++;
                (*output)[i+1][j] = 0;
            }
            if(mergeCounter != 0 && (*output)[i+1][j] != 0){
                (*output)[i][j] = (*output)[i+1][j];
                (*output)[i+1][j] = 0;
            }
        }
        mergeCounter = 0;
    }

}



__device__ void downSolver(Board * output){
    int8_t i, j, moveCounter, mergeCounter;

    //This section moves all items through the 0's.
    //Might not need to dereference board pointers
    moveCounter = 0;
    for(j=0; j < WIDTH; j++){
        for(i=HEIGHT-1; i >= 0; i--){
            if((*output)[i][j] == 0){
                moveCounter++;
            }
            else if(moveCounter != 0){
                (*output)[(i+moveCounter)][j] = (*output)[i][j];
                (*output)[i][j] = 0;
            }
        }
        moveCounter = 0;
    }
    mergeCounter = 0;
    for(j=0; j < WIDTH; j++){
        for(i=HEIGHT-1; i > 0; i--){
            if((*output)[i][j] == (*output)[i-1][j]){
                (*output)[i][j] = 2*((*output)[i][j]);
                mergeCounter++;
                (*output)[i-1][j] = 0;
            }
            if(mergeCounter != 0 && (*output)[i-1][j] != 0){
                (*output)[i][j] = (*output)[i-1][j];
                (*output)[i-1][j] = 0;
            }
        }
        mergeCounter = 0;
    }

}

/**
 * This function adds the random move to the board. This will most likely change later on to fit with the CUDA program so they produce the same results.
 * @param movedBoard A pointer to a Board object to have a random tile added to the board.
 */
 __device__ void randGen(Board * movedBoard){
    unsigned long long seed= (*movedBoard)[0][0] + 2 * (*movedBoard)[0][1] + 3 * (*movedBoard)[0][2] + 4 * (*movedBoard)[0][3] + 5 * (*movedBoard)[1][0] + 6 * (*movedBoard)[1][1] + 7 * (*movedBoard)[1][2] + 8 * (*movedBoard)[1][3];
    hiprandState_t *state;
    hiprand_init(seed, 0,0, state);

    unsigned int randNum = hiprand(state);
    unsigned char position = randNum % SIZE;
    while((*movedBoard)[(position/WIDTH)][(position%HEIGHT)] != 0){
        randNum = hiprand(state);
        position = randNum % SIZE;
    }
    unsigned int randomValue = hiprand(state);
    if(randomValue % 10 == 9){
        (*movedBoard)[(position/WIDTH)][(position%HEIGHT)] = 4;
    }
    else{
        (*movedBoard)[(position/WIDTH)][(position%HEIGHT)] = 2;
    }


}


/**
 *This takes the predetermined move and returns a Board that has had that move applied. This should be the link between the recursive section of the code and the solver
 * @param input The board that is requested to be solved
 * @param currMove The move to apply to the board
 * @param output A pointer for the board after the move has occurred to be stored in.
 * @return Returns the status of the move. Whether or not the board was updated.
 */
__device__ status moveHandler(Board *input, Move currMove){

    

    switch(currMove){
        case(up):
            // printf("Moving up \r\n");
            upSolver(input);
            break;
        case down:
            // printf("Moving down \r\n");
            downSolver(input);
            break;
        case left:
            // printf("Moving left \r\n");
            leftSolver(input);
            break;
        case right:
            // printf("Moving right \r\n");
            rightSolver(input);
            break;

    }

    bool changed = false;
    bool fail = true;

    for(uint8_t i=0; i < HEIGHT; i++){
        for(uint8_t j=0; j < WIDTH;j++) {
            if ((*input)[i][j] == 0) {
                fail = false;
            }
            //if ((*checkBoard)[i][j] != (*input)[i][j]) {
            //    changed = true;
            //}
        }
    }

    if(fail){
        return boardFull;
    }
    else if(!changed){
        return boardUnchanged;
    }
    randGen(input);
    return boardUpdated;

}




__global__ void kernel(Board *BoardIn, int * scoreList){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bd = blockDim.x;

    uint32_t threadNum = bx * bd + tx;

    Board board;
    int i,j;
    for(i = 0; i < HEIGHT; i++){
        for(j = 0; j < WIDTH; j++){
            board[i][j] = (BoardIn)[i][j];
        }
    }

    status stat;
    Move mList[NUMMOVES];

    //Bitwise and with mask ends up creating many invalid moves (see: 0xC000 & 0xC000), need to rightshift
    mList[0] = (Move) (threadNum & m0Mask);
    mList[1] = (Move) (threadNum & m1Mask);
    mList[2] = (Move) (threadNum & m2Mask);
    mList[3] = (Move) (threadNum & m3Mask);
    mList[4] = (Move) (threadNum & m4Mask);
    mList[5] = (Move) (threadNum & m5Mask);
    mList[6] = (Move) (threadNum & m6Mask);
    mList[7] = (Move) (threadNum & m7Mask);

    scoreList[threadNum] = 0;
    for(i = 0; i < NUMMOVES; i++){
        stat = moveHandler(&board,mList[i]);
        if(stat != boardUpdated){
            break;
        }
        if(i != 7){
            randGen(&board);
        }
        else{
            scoreList[threadNum] = score(&board);
        }
        if(threadNum == 0){
            printf("LIST %d: ",i);
            for(int q = 0; i < HEIGHT; i++){
                for(int r = 0; j < WIDTH; j++){
                    printf("%d ",board[q][r]);
                }
            }
            printf("\r\n");
        }


    }
    if(scoreList[threadNum] != 0){
        printf("DEBUG SCORE:%d\r\n",scoreList[threadNum]);
    }
    __syncthreads();
    return;
}




int main(int argc, char **argv) {
    wbArg_t arg;
    Board hostInputBoard;
    Board * deviceInputBoard;
    char *inputBoardFile;
    int *hostScoreList;
    int *deviceScoreList;
    int Score;
    int inputLength;
        
    arg = wbArg_read(argc, argv);
    
    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

    wbTime_start(GPU, "Doing GPU memory allocation");


    int numScores = (int)pow(4, 8);
    int scoreListSize = numScores * sizeof(int);
    int boardSize = SIZE * sizeof(int);
    
    //inputBoardFile = (char *)wbImport(wbArg_getInputFile(arg, 0), &inputLength);
    hostScoreList = (int *)malloc(scoreListSize);
    /*
    for(int i = 0; i < HEIGHT; i++){
        for(int j = 0; j < WIDTH; j++){
            (*hostInputBoard)[i][j] = (*inputBoardFile)[i * WIDTH + j];
        }
    }
    */

    
    hostInputBoard[0][0] = 0;
    hostInputBoard[0][1] = 0;
    hostInputBoard[0][2] = 0;
    hostInputBoard[0][3] = 0;
    hostInputBoard[1][0] = 2;
    hostInputBoard[1][1] = 0;
    hostInputBoard[1][2] = 0;
    hostInputBoard[1][3] = 0;
    hostInputBoard[2][0] = 0;
    hostInputBoard[2][1] = 2;
    hostInputBoard[2][2] = 0;
    hostInputBoard[2][3] = 0;
    hostInputBoard[3][0] = 0;
    hostInputBoard[3][1] = 0;
    hostInputBoard[3][2] = 0;
    hostInputBoard[3][3] = 0;
    

    wbCheck(hipMalloc((void**)&deviceScoreList, scoreListSize)); 
    wbCheck(hipMalloc((void**)&deviceInputBoard, boardSize)); 

    wbTime_stop(GPU, "Doing GPU memory allocation");

    wbTime_start(Copy, "Copying data to the GPU");
    
    wbCheck(hipMemcpy(deviceInputBoard, &hostInputBoard, boardSize, hipMemcpyHostToDevice));


    wbTime_stop(Copy, "Copying data to the GPU");

    wbTime_start(Compute, "Doing the computation on the GPU");

    dim3 DimGrid(256, 1, 1);
    dim3 DimBlock(256, 1, 1);
    kernel<<<DimGrid, DimBlock>>>(deviceInputBoard,deviceScoreList);

    wbTime_stop(Compute, "Doing the computation on the GPU");
    
    hipDeviceSynchronize();
    wbCheck(hipPeekAtLastError());
    ////////////////////////////////////////////////////
    wbTime_start(Copy, "Copying data from the GPU");
    wbCheck(hipMemcpy(hostScoreList, deviceScoreList, scoreListSize, hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying data from the GPU");

    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    for(uint32_t num = 0; num < numScores; num++){\
        if(hostScoreList[num] != 0){
            printf("Score %d is %d \r\n", num, hostScoreList[num]);
        }
    }

    wbSolution(arg, hostScoreList, scoreListSize);

    wbCheck(hipFree(deviceScoreList));

    return 0;
}
